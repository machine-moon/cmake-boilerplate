
#include <hip/hip_runtime.h>
#ifdef TWIZ_GPU_RUNTIME_HIP
#include "Examples/gpu.h"
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <tuple>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

#define HIP_ASSERT(c) {                               \
   if (c != hipSuccess){                              \
      printf("HIP Error : %s", hipGetErrorString(c)); \
      printf(" %s %d\n", __FILE__, __LINE__);         \
      exit(c);                                        \
   }                                                  \
}


namespace Twiz
{
    __global__ void vecAdd(const float* pa, const float* pb, float* pc, int size)
    {
        int i = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (i < size)
        {
            pc[i] = pa[i] * pb[i];
        }
    }

    void VectorAdd(const float* pda, const float* pdb, float* pdc, int n)
    {
        float *da = nullptr, *db = nullptr, *dc = nullptr;
        HIP_ASSERT(hipMalloc(&da, n * sizeof(float)));
        HIP_ASSERT(hipMalloc(&db, n * sizeof(float)));
        HIP_ASSERT(hipMalloc(&dc, n * sizeof(float)));

        HIP_ASSERT(hipMemcpy(da, pda, n * sizeof(float), hipMemcpyHostToDevice));
        HIP_ASSERT(hipMemcpy(db, pdb, n * sizeof(float), hipMemcpyHostToDevice));

        int blockSize = 256;
        int gridSize = (n + blockSize - 1) / blockSize;
        vecAdd<<<gridSize, blockSize>>>(da, db, dc, n);
        HIP_ASSERT(hipDeviceSynchronize());

        HIP_ASSERT(hipMemcpy(pdc, dc, n * sizeof(float), hipMemcpyDeviceToHost));

        HIP_ASSERT(hipFree(da));
        HIP_ASSERT(hipFree(db));
        HIP_ASSERT(hipFree(dc));
    }

    std::tuple<float*, float*, float*> Generate(int size)
    {
        float* a = new float[size];
        float* b = new float[size];
        float* c = new float[size];

        for (int i = 0; i < size; ++i)
        {
            a[i] = static_cast<float>(i);
            b[i] = static_cast<float>(3 * i);
            c[i] = 0.0f;
        }

        return std::make_tuple(a, b, c);
    }

    void Cleanup(float* pa, float* pb, float* pc)
    {
        delete[] pa;
        delete[] pb;
        delete[] pc;
    }
} // namespace Twiz
#endif // TWIZ_GPU_RUNTIME_HIP