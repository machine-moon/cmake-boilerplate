
#include <hip/hip_runtime.h>
#ifdef TWIZ_GPU_RUNTIME_CUDA
#include "Examples/gpu.h"
#include <cstdio>
#include <cstdlib>
#include <cuda_runtime.h>
#include <tuple>

#define CUDA_ASSERT(x)                                                                                                                                                                                 \
    do                                                                                                                                                                                                 \
    {                                                                                                                                                                                                  \
        cudaError_t result = (x);                                                                                                                                                                      \
        if (result != cudaSuccess)                                                                                                                                                                     \
        {                                                                                                                                                                                              \
            printf("CUDA Error: %s at %s:%d\n", cudaGetErrorString(result), __FILE__, __LINE__);                                                                                                       \
            exit(1);                                                                                                                                                                                   \
        }                                                                                                                                                                                              \
    } while (0)

namespace Twiz
{
    __global__ void vecAdd(const float* pa, const float* pb, float* pc, int size)
    {
        int i = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (i < size)
        {
            pc[i] = pa[i] + pb[i];
        }
    }

    void VectorAdd(const float* pda, const float* pdb, float* pdc, int n)
    {
        float *da = nullptr, *db = nullptr, *dc = nullptr;
        CUDA_ASSERT(cudaMalloc(&da, n * sizeof(float)));
        CUDA_ASSERT(cudaMalloc(&db, n * sizeof(float)));
        CUDA_ASSERT(cudaMalloc(&dc, n * sizeof(float)));

        CUDA_ASSERT(cudaMemcpy(da, pda, n * sizeof(float), cudaMemcpyHostToDevice));
        CUDA_ASSERT(cudaMemcpy(db, pdb, n * sizeof(float), cudaMemcpyHostToDevice));

        int blockSize = 256;
        int gridSize = (n + blockSize - 1) / blockSize;
        vecAdd<<<gridSize, blockSize>>>(da, db, dc, n);
        CUDA_ASSERT(cudaDeviceSynchronize());

        CUDA_ASSERT(cudaMemcpy(pdc, dc, n * sizeof(float), cudaMemcpyDeviceToHost));

        CUDA_ASSERT(cudaFree(da));
        CUDA_ASSERT(cudaFree(db));
        CUDA_ASSERT(cudaFree(dc));
    }

    std::tuple<float*, float*, float*> Generate(int size)
    {
        float* a = new float[size];
        float* b = new float[size];
        float* c = new float[size];

        for (int i = 0; i < size; ++i)
        {
            a[i] = static_cast<float>(i);
            b[i] = static_cast<float>(2 * i);
            c[i] = 0.0f;
        }

        return std::make_tuple(a, b, c);
    }

    void Cleanup(float* pa, float* pb, float* pc)
    {
        delete[] pa;
        delete[] pb;
        delete[] pc;
    }
} // namespace Twiz
#endif // _GPU_RUNTIME_CUDA
